
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <ctime>
#include <chrono>

//----------------------------------------------------------------
__global__
void
add(const float *const a, const float *const b, float *const c)
{
    unsigned int index = threadIdx.x;

    c[index] = a[index] + b[index];
}


//----------------------------------------------------------------
int
main(int argc, char **argv)
{
    hipError_t cu_err;
    const unsigned int mem_size = sizeof(float) * 1024;
    float *managed_a, *managed_b, *managed_c;

    cu_err = hipMallocManaged(&managed_a, mem_size);
    assert(cu_err == hipSuccess);

    cu_err = hipMallocManaged(&managed_b, mem_size);
    assert(cu_err == hipSuccess);

    cu_err = hipMallocManaged(&managed_c, mem_size);
    assert(cu_err == hipSuccess);

    for( unsigned i = 0 ; i < 1024 ; i++ ) {
        managed_a[i] = (float)i;
        managed_b[i] = (1024 - i);
    }

    auto start_time = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 100000; ++i) {
        add<<<1, 1024>>>(managed_a, managed_b, managed_c);
    }
    hipDeviceSynchronize();

    auto end_time = std::chrono::high_resolution_clock::now();
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

    std::cout << ms.count() << "ms" << std::endl;

    for( unsigned i = 0 ; i < 1024 ; i++ ) {
        if ( managed_c[i] != 1024.0 ) {
            std::cerr << "error:" << i << " " << managed_c[i] << std::endl;
        }
    }

    hipFree(managed_a);
    hipFree(managed_b);
    hipFree(managed_c);
}
