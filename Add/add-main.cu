
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <ctime>
#include <chrono>

//----------------------------------------------------------------
__global__
void
add(const float *const a, const float *const b, float *const c)
{
    unsigned int x = threadIdx.x;
    unsigned int y = threadIdx.y;
    unsigned int index = y * blockDim.x + x;

    c[index] = a[index] + b[index];
}


//----------------------------------------------------------------
int
main(int argc, char **argv)
{
    hipError_t cu_err;
    const unsigned int mem_size = sizeof(float) * 1024;
    float *managed_a, *managed_b, *managed_c;

    cu_err = hipMallocManaged(&managed_a, mem_size);
    assert(cu_err == hipSuccess);

    cu_err = hipMallocManaged(&managed_b, mem_size);
    assert(cu_err == hipSuccess);

    cu_err = hipMallocManaged(&managed_c, mem_size);
    assert(cu_err == hipSuccess);

    auto start_time = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 100000; ++i) {
        add<<<1, 1024>>>(managed_a, managed_b, managed_c);
    }
    hipDeviceSynchronize();

    auto end_time = std::chrono::high_resolution_clock::now();
    auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

    std::cout << ms.count() << "ms" << std::endl;

    hipFree(managed_a);
    hipFree(managed_b);
    hipFree(managed_c);
}
